
#include <hip/hip_runtime.h>
#ifdef CUDA_ON
#include <dp.cuh>

namespace DP {
  namespace {
    class Cell {
    private:
      Angle _th;   ///<Angle of the final point of the point to point curve.
      LEN_T _l;    ///<Length of the point to point curve.
      int _nextID; ///<Id to the next cell for dynamic programming.

    public:
      /*!
       * Default void constructor which returns a cell initialized with ANGLE::FREE, max length and -1 as next cell.
       */
      Cell() : _th(ANGLE::FREE), _l(MAX_LEN_T), _nextID(0) {}

      /*!
       * Constructor that takes in input an angle, a length and the next id and returns a DP::Cell.
       * @param th The initial angle of the point to point curve.
       * @param l The length of the point to point curve.
       * @param next The next id of the cell.
       */
      BOTH Cell(Angle th, LEN_T l, int nextID) :
          _th(th), _l(l), _nextID(nextID) {}

      BOTH Angle th() const { return this->_th; }      ///<Returns the angle.
      BOTH LEN_T l()  const { return this->_l; }       ///<Returns the length.
      BOTH int next() const { return this->_nextID; }  ///<Returns the next id.

      /*!
       * Sets the new angle.
       * @param th The new angle to be set.
       * @return the new set angle.
       */
      BOTH Angle th(Angle th) {
        this->_th = th;
        return this->th();
      }
      /*!
       * Sets the new length.
       * @param th The new length to be set.
       * @return the new set length.
       */
      BOTH LEN_T l(LEN_T l) {
        this->_l = l;
        return this->l();
      }
      /*!
       * Sets the new next id.
       * @param th The new next id to be set.
       * @return the new set next id.
       */
      BOTH int next(int nextID){
        //printf("nextID in class %d %u\n", nextID, nextID);
        this->_nextID = nextID;
        return this->next();
      }

      /*!
       * Creates a deep copy of a cell to `this`.
       * @param d The cell to copy from.
       * @return `*this`.
       */
      BOTH Cell& copy(const Cell &d) {
        this->th(d.th());
        this->l(d.l());
        this->next(d.next());

        return *this;
      }
      /*!
       * Overrides the assign operator (=) to make a deep copy of a cell to `this`.
       * @param d The cell to copy from.
       * @return `*this`.
       */
      BOTH Cell& operator=(const Cell &d) {
        this->copy(d);
        return *this;
      }

      /*!
       * Function to print the most essential info about `DP::Cell`.
       * @param pretty An additional truth value to print a prettier version. Default is `false`.
       * @return A `std::stringstream` object containing the data of `DP::Cell`.
       */
      std::stringstream to_string(bool pretty = false) const {
        std::stringstream out;
        out << std::setw(20) << std::setprecision(17);
        if (pretty) {
          out << "th: " << this->th() << " l: " << this->l();
        } else {
          out << "<" << (Angle)(this->th()*1.0) << ", " << (LEN_T)(this->l()) << ">";
        }
        return out;
      }
      /*! This function overrides the << operator so to print with `std::cout` the most essential info about the `DP::Cell`.
          \param[in] out The out stream.
          \param[in] data The `DP::Cell` to print.
          \returns An output stream to be printed.
      */
      friend std::ostream &operator<<(std::ostream &out, const Cell &data) {
        out << data.to_string().str();
        return out;
      }

    };
  } //Anonymous namespace to hide information
} //Namespace DP

__global__ void dubinsWrapper(Configuration2 c0, Configuration2 c1, double Kmax, double* L){
  CURVE c(c0, c1, Kmax);
  //printf("%.17f\n", c.l());
  L[0]+=c.l();
}

__global__ void printResults(real_type* results, uint discr, uint size){
  for (int i=0; i<size; i++){
    for(int j=0; j<discr; j++){
      for(int h=0; h<discr; h++){
        printf("(%2.0f,%.2f)", (float)((i*discr+j)*discr+h), results[(i*discr+j)*discr+h]);
      }
      printf("\t");
    }
    printf("\n");
  }
}

__global__ void printMatrix(DP::Cell* matrix, uint discr, uint size){
  for (int i=0; i<size; i++){
    for(int j=0; j<discr; j++){
      printf("(%d,%d)", (i*discr+j), matrix[i*discr+j].next());
    }
    printf("\n");
  }
}


// returns (up to) two circles through two points, given the radius
// Marco Frego and Paolo Bevilacqua in "An Iterative Dynamic Programming Approach to the Multipoint Markov-Dubins Problem" 2020.
static inline
void circles(real_type x1, real_type y1, real_type x2, real_type y2, real_type r, std::vector<real_type> & XC, std::vector<real_type> & YC) 
{
  real_type TOL = 1e-8;
  
  real_type q = std::hypot(x2-x1, y2-y1);
  real_type x3 = 0.5*(x1+x2);
  real_type y3 = 0.5*(y1+y2);

  real_type delta = r*r-q*q/4.;
    
  XC.clear();
  YC.clear();

  if (delta < -TOL) {
    return;
  }
  
  if (delta < TOL) 
  {
    XC.push_back(x3);
    YC.push_back(y3);
  }
  else
  {
    real_type deltaS = std::sqrt(delta);
    XC.push_back(x3 + deltaS*(y1-y2)/q);
    YC.push_back(y3 + deltaS*(x2-x1)/q);
    XC.push_back(x3 - deltaS*(y1-y2)/q);
    YC.push_back(y3 - deltaS*(x2-x1)/q);
  }
}

// Marco Frego and Paolo Bevilacqua in "An Iterative Dynamic Programming Approach to the Multipoint Markov-Dubins Problem" 2020.
// The function name is pretty self-explainatory 
uint guessInitialAngles(std::vector<std::set<Angle> >& moreAngles, const std::vector<Configuration2>& points, const std::vector<bool> fixedAngles, const real_type K){
  uint max=0;
  for (uint i=1; i<points.size(); i++){
    moreAngles.push_back(std::set<Angle>());
    if (i==1) { moreAngles.push_back(std::set<Angle>()); }
    //First add the lines connecting two points:
    Angle th = std::atan2((points[i].y()-points[i-1].y()), (points[i].x()-points[i-1].x()));
    if (!fixedAngles[i-1]){ moreAngles[i-1].insert(th); }
    if (!fixedAngles[i])  { moreAngles[i].insert(th); }
    
    //Then add the possible angles of the tangents to two possible circles:
    std::vector<real_type> XC, YC;
    circles(points[i-1].x(), points[i-1].y(), points[i].x(), points[i].y(), 1./K, XC, YC);
    
    for (uint j=0; j<XC.size(); j++){
      if (!fixedAngles[i-1]){
        th = std::atan2(points[i-1].y()-YC[j], points[i-1].x()-XC[j]);
        moreAngles[i-1].insert(th+M_PI/2.);
        moreAngles[i-1].insert(th-M_PI/2.);
      }
      if (!fixedAngles[i]){
        th = std::atan2(points[i].y()-YC[j], points[i].x()-XC[j]);
        moreAngles[i].insert(th+M_PI/2.);
        moreAngles[i].insert(th-M_PI/2.);
      }
    }
    if (moreAngles[i-1].size()>max){
      max=moreAngles[i-1].size();
    }
    if (i==points.size()-1 && moreAngles[i].size()>max){
      max=moreAngles[i].size();
    }
  }  
  return max;
}

std::vector<Angle> bestAngles(DP::Cell* matrix, int discr, int size){
  DP::Cell* best=&matrix[0];
  //Find best path
  for (int i=size; i<discr*size; i+=size){
    if (best->l()>matrix[i].l()  && matrix[i].l()!=0){ //TODO The second check is actually a bug in solveCell, but I'm not in the right mind to find this bug, please fix later
      best=&matrix[i];
    }
  }
  //Retrieve best angles
  std::vector<Angle> ret(1, best->th());
  uint nextID=best->next();
  while (nextID!=0){
    ret.push_back(matrix[nextID].th());
    nextID=matrix[nextID].next();
  }
  return ret;
}

std::pair<LEN_T, std::vector<Angle> >
bestAnglesMatrix(DP::Cell* matrix, int discr, int size, const std::vector<bool>& fixedAngles){
  DP::Cell* best=&matrix[0];

  if (!fixedAngles[0]){
    for(int i=1; i<discr; i++){
      if (matrix[i].l()<best->l())
        best=&matrix[i];
    }
  }

  //std::cout << "In function Length: " << std::setw(20) << std::setprecision(17) << best->l() << std::endl;

  std::vector<real_type> ret={best->th()};
//  std::vector<real_type> ret={best->l(), best->th()};
  int nextID=best->next()+discr;
  for (int i=1; i<size; i++){
    ret.push_back(matrix[nextID].th());
    nextID=matrix[nextID].next()+(i+1)*discr;
  }
  //ret.insert(ret.begin(), best->l());
  return std::pair<LEN_T, std::vector<Angle> >(best->l(), ret);
}

__global__ void solveCol( DP::Cell* matrix, uint discr, uint size, const bool* fixedAngles, 
                          Configuration2 c0, Configuration2 c1, 
                          Angle a00, Angle a01, real_type* params, int i, Angle fullAngle, bool halveDiscr
                        ){
  int tidx=threadIdx.x+blockDim.x*blockIdx.x;
  int stride=blockDim.x*gridDim.x;
  int halfDiscr=(discr-1)/2;
  int j=tidx;

  // if (j<discr){
  for (; j<discr; j+=stride){
    Angle bestA=0.0;
    LEN_T bestL=MAX_LEN_T; 
    int bestK=0;
    if (!fixedAngles[i-1]){ //If angle is fixed I don't have to change it
      double hj=fullAngle*((j-halfDiscr)*1.0)/(((halveDiscr ? halfDiscr : discr)*1.0));
      c0.th(a00+hj); 
    } 
    
    for (int k=0; k<discr; k++){ //SolveCell
      LEN_T currL=MAX_LEN_T;
      if (!fixedAngles[i]){ //If angle is fixed I don't have to change its
        double hk=fullAngle*((k-halfDiscr)*1.0)/(((halveDiscr ? halfDiscr : discr)*1.0));
        c1.th(a01+hk); 
      } 
      CURVE c=CURVE(c0, c1, params); 
      DP::Cell* next=(i==size-1 ? NULL : &matrix[k*size+(i+1)]);
      if (c.l()>0){
        currL=c.l();
        if (next!=NULL){
          currL+=next->l();
        }  
        if (currL<bestL || bestL==MAX_LEN_T){
          bestL=currL;
          bestA=c1.th();
          bestK=k;
        }
      }
      if (fixedAngles[i]){ k=discr; } //If the angle is fixed I don't have to change it
    }
    
    if (bestL!=MAX_LEN_T){
      uint nextID=(i==size-1 ? 0 : bestK*size+(i+1));
      matrix[j*size+i]=DP::Cell(bestA, bestL, nextID);
    }
    if (i==1){
      matrix[size*j]=DP::Cell(c0.th(), bestL, (size*j+i));
    }
    if(fixedAngles[i-1]) j=discr;
  }
}


__global__ void solveMatrixCol (DP::Cell* matrix, uint discr, uint size, const bool* fixedAngles, 
                                Configuration2 c0, Configuration2 c1, 
                                real_type* params, int i, uint ref=0){
  uint tidx=threadIdx.x+blockDim.x*blockIdx.x;
  uint stride=blockDim.x*gridDim.x;

  uint j=tidx;
  // if (j<discr){
  for (; j<discr; j+=stride){
    c0.th(matrix[i*discr+j].th());
    for (int h=0; h<(int)(discr); h++){
      c1.th(matrix[(i+1)*discr+h].th());

      CURVE c=CURVE(c0, c1, params);
      LEN_T currL=c.l()+matrix[(i+1)*discr+h].l();
      //if (ref==3 && i==0 && j==0){
      //  printf("x0: %.2f y0: %.2f th0: %.16f x1: %.2f y1: %.2f th1: %.16f matrix[i*discr+j].l(): %.16f currL %.16f c.l(): %.16f matrix[(i+1)*discr+h].l(): %.16f\n", c0.x(), c0.y(), c0.th(), c1.x(), c1.y(), c1.th(), (matrix[i*discr+j].l()<10000.0 ? matrix[i*discr+j].l() : 10000.0), currL, c.l(), matrix[(i+1)*discr+h].l());
      //}
      if (currL<matrix[i*discr+j].l()) {
        matrix[i*discr+j].l(currL);
        //printf("nextID in func: %u %d\n", h, h);
        matrix[i*discr+j].next(h);
      }
      if (fixedAngles[i+1]) {h=discr;}
    }
    if (matrix[i*discr+j].next()==-1) printf("[%u] BIG NO\n", i*discr+j);
    if (fixedAngles[i]) {j=discr;}
  }
}

void solveDPMatrix (std::vector<Configuration2> points, DP::Cell* dev_matrix, uint discr, std::vector<bool> fixedAngles, 
                    bool* dev_fixedAngles, real_type* dev_params, uint nThreads=128, uint ref=0){

  //REMOVE
  size_t size=points.size();
  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());


  for (int i=size-2; i>=0; i--){
    Configuration2 c0=points[i];
    Configuration2 c1=points[i+1];

    size_t threads=discr>nThreads ? nThreads : discr;
    size_t blocks=((int)(discr/threads)+1)*numberOfSMs; 
    
    if(fixedAngles[i]){
      threads=1;
      blocks=1;
    }
    solveMatrixCol<<<blocks, threads>>>(dev_matrix, discr, size, dev_fixedAngles, c0, c1, dev_params, i, ref);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError()); 
  }
  if (ref==30){
    printMatrix<<<1,1>>>(dev_matrix, discr, size);
    cudaDeviceSynchronize();
  }
}

std::pair<LEN_T, std::vector<Angle> >
solveDPMatrixAllocator (std::vector<Configuration2> points, uint discr,
                        const std::vector<bool> fixedAngles, std::vector<real_type> params,
                        Angle fullAngle=2*M_PI, uint nThreads=0, uint ref=0){
  size_t size=points.size();
  DP::Cell* matrix;
  bool* dev_fixedAngles=cudaSTDVectorToArray<bool>(fixedAngles);
  real_type* dev_params=cudaSTDVectorToArray<real_type>(params);  
  DP::Cell* dev_matrix;

  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());
  
  std::vector<std::set<Angle> > moreAngles;
  uint addedAngles=0;

  addedAngles=guessInitialAngles(moreAngles, points, fixedAngles, params[0]);

  uint halfDiscr=(uint)((discr-(discr%2==0 ? 0 : 1))/2);
  real_type dtheta=fullAngle/(((int)(discr/2))*1.0);
  if(ref==0){
    dtheta=fullAngle/(discr*1.0);
  }
  discr+=addedAngles;
  cudaMallocHost(&matrix, (sizeof(DP::Cell)*size*discr));
  cudaMalloc(&dev_matrix, sizeof(DP::Cell)*size*discr);

  for (uint i=0; i<size; i++){ //TODO change back, remove l=-1 if fixedAngles
    LEN_T l = (i==size-1 ? 0 : std::numeric_limits<LEN_T>::max());
    for (uint j=0; j<=halfDiscr; j++){
      if (fixedAngles[i]){
        matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1);
        break;
      }
      else {
        if(j==0) { 
          matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1); 
        }
        else{
          matrix[i*discr+j]          =DP::Cell(mod2pi(points[i].th()-(j*1.0)*dtheta), l, -1);
          matrix[i*discr+j+halfDiscr]=DP::Cell(mod2pi(points[i].th()+(j*1.0)*dtheta), l, -1); 
        }
      }
    }
    if (true){
      uint j=discr-addedAngles;
      if (!fixedAngles[i]){
        for (std::set<Angle>::iterator it=moreAngles[i].begin(); it!=moreAngles[i].end(); ++it){
          matrix[i*discr+j]=DP::Cell(*it, l, -1);
          j++;
        }
        for (; j<discr; j++){
          matrix[i*discr+j]=DP::Cell(0, l, -1);
        }
      }
    }
  }
    
  cudaMemcpy(dev_matrix, matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyHostToDevice);
  cudaCheckError(cudaGetLastError());

  solveDPMatrix(points, dev_matrix, discr, fixedAngles, dev_fixedAngles, dev_params, nThreads, ref);

  cudaMemcpy(matrix, dev_matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyDeviceToHost);
  cudaCheckError(cudaGetLastError());


#ifdef DEBUG
  //Retrieve angles
  cout << "Computing best angles" << endl;
#endif
  std::pair<LEN_T, std::vector<Angle> > ret=bestAnglesMatrix(matrix, discr, size, fixedAngles);
#ifdef DEBUG
  std::vector<Angle> bestA=ret.second;
  printV(bestA)
#endif
  
#ifdef DEBUG
  LEN_T Length=0.0;
  for (unsigned int i=bestA.size()-1; i>0; i--){
    points[i].th(bestA[i]);
    points[i-1].th(bestA[i-1]);
    CURVE c(points[i-1], points[i], params.data());
    Length+=c.l();
  }
  cout << "\tMatrix length: " << setprecision(20) << Length << " " << setprecision(12) << (Length-7.46756219733842652175326293218) << endl;
  cout << "Printing for Matlab" << endl;
  cout << "X=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].x() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "Y=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].y() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "th=[";
  for (unsigned int i=0; i<bestA.size(); i++){ cout << bestA[i] << (i!=bestA.size()-1 ? ", " : "];\n"); }
  cout << "KMAX: " << params[0] << endl;
#endif

  cudaFreeHost(matrix);

  cudaFree(dev_matrix);
  cudaFree(dev_params);
  cudaFree(dev_fixedAngles);

  return ret;
}


__global__ void computeMore(DP::Cell* matrix, real_type* results, const bool* fixedAngles,
                            real_type* params, const Configuration2* points, 
                            size_t jump, size_t discr, size_t size, size_t iter){ //TODO It may be possible to remove cmp and use iter and i to compute the position
  uint tidx=threadIdx.x+blockDim.x*blockIdx.x;

  uint j=tidx;
  if (j<discr*jump*discr){        //j must be less than the number of rows (jump) times the number of inner cells per cell, times the number of cells per row
    uint cell=(int)(tidx/discr);  //The big cell
    uint inCell=tidx%discr;       //The small cell inside the big cell
    uint cmpId=(int)(cell/discr); //The row in this call
    uint pos=iter*jump+cmpId;     //The row w.r.t. the whole matrix

    if (pos<size-1){ 
      Configuration2 c0=points[pos];
      Configuration2 c1=points[pos+1];

      if (!fixedAngles[pos])   {c0.th(matrix[cell+iter*jump*discr].th());}
      if (!fixedAngles[pos+1]) {c1.th(matrix[inCell+(pos+1)*discr].th());}

      CURVE c=CURVE(c0, c1, params);
      if (c.l()>0){
        results[cell*discr+inCell]=c.l();
      }
    }
  }
}


void bestAnglesPerCell( DP::Cell* matrix, real_type* results, const std::vector<bool> fixedAngles, 
                        size_t size, size_t discr, size_t iter, size_t jump, size_t _threads, size_t numberOfSMs){
  //MIND THAT WE START FROM THE LAST ROW OF THE MATRIX
  int startRowIDM=iter*jump;        //Given #jump rows, this is the id of the first row in the jump group
  uint lastRowIDM=iter*jump+jump-1;  //Given #jump rows, this is the id of the last row in the jump group
  lastRowIDM=(lastRowIDM<size ? lastRowIDM : size-1);
  for (int i=lastRowIDM; i>=startRowIDM; i--){ //Cycle through the rows
    if (i==(int)(size-1)){continue;}    //If it's the last row, then skip it.
    uint startCellM=i*discr;              //Given #jump rows, this is the id of the cell in the row in the jump group I'm considering
    //std::cout << "startRowIDM: " << startRowIDM << std::endl;
    //std::cout << "lastRowIDM: " << lastRowIDM << std::endl;
    //std::cout << "startCellM: " << startCellM << std::endl;
    //std::cout << "i: " << i << std::endl;
    for (uint cellIDM=startCellM; cellIDM<startCellM+discr; cellIDM++){ //Cycle through all the cells in the row  
      for (uint h=0; h<discr; h++){        //Each cell in matrix corresponds to discr cells in results. Each h is in results is the same as the next row
        uint cellIDR=cellIDM*discr+h-startRowIDM*discr*discr;
        double currL=results[cellIDR]+matrix[(i+1)*discr+h].l();
        // int a=-2;
        if(currL<matrix[cellIDM].l()){
          matrix[cellIDM].l(currL);
          matrix[cellIDM].next(h);
          // a=matrix[cellIDM].next(h);
          //std::cout << "a: " << a << std::endl;
        }
        //if (cellIDM>29 && cellIDM<45){
        //  std::cout << "i: " << i << std::endl;
        //  std::cout << "cellIDM: " << cellIDM << std::endl;
        //  std::cout << "cellIDR: " << cellIDR << std::endl;
        //  std::cout << "results[cellIDR]: " << results[cellIDR] << std::endl;
        //  std::cout << "currL: " << currL << std::endl;
        //  std::cout << "matrix[cellIDM].l(): " << matrix[cellIDM].l() << std::endl;
        //  std::cout << "a: " << a << std::endl;
        //  std::cout << "matrix[cellIDM].n(): " << matrix[cellIDM].next() << std::endl;
        //}
        //std::cout << "matrix[(i+1)*discr+h].l(): " << matrix[(i+1)*discr+h].l() << std::endl;
        if (fixedAngles[i+1]){ h=discr; }
      } 
      if (matrix[cellIDM].next()<0) {printf("[%u] BIG NO\n", cellIDM);}
    }
    if (fixedAngles[i]){ i=startRowIDM-1; }
  }
}

std::pair<LEN_T, std::vector<Angle> >
solveDPAllIn1 ( std::vector<Configuration2> points, uint discr, const std::vector<bool> fixedAngles,
                std::vector<real_type> params, Angle fullAngle, uint nThreads=0, uint ref=0){

  //Get the number of multiproccessors in the GPU so to best compute the number of blocks afterwards.
  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());


  uint addedAngles=0;
  std::vector<std::set<Angle> > moreAngles;
  addedAngles=guessInitialAngles(moreAngles, points, fixedAngles, params[0]);

  size_t size=points.size();
  //discr=(discr%2==0 ? discr+1 : discr); //So.... since we add always the angle in position 0, we'll always have an odd number of discretizionations... I'm not so sure about this, but ok
  uint halfDiscr=(uint)(discr/2);
  real_type dtheta=fullAngle/(((int)(discr/2))*1.0);
  if(ref==0){
    dtheta=fullAngle/(discr*1.0);
  }
  discr+=addedAngles;

  DP::Cell* matrix;
  cudaMallocHost(&matrix, sizeof(DP::Cell)*size*(discr));
  DP::Cell* dev_matrix;
  cudaMalloc(&dev_matrix, sizeof(DP::Cell)*size*(discr));

  bool* dev_fixedAngles=cudaSTDVectorToArray<bool>(fixedAngles);
  real_type* dev_params=cudaSTDVectorToArray<real_type>(params);  
  Configuration2* dev_points=cudaSTDVectorToArray<Configuration2>(points);

  for (uint i=0; i<size; i++){
    LEN_T l = (i==size-1 ? 0 : std::numeric_limits<LEN_T>::max());
    if (fixedAngles[i]){
      for (uint j=0; j<discr; j++){
        matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1);
        //In this case I need to have the row full of the same values. Otherwise I should change the kernel function and add particular cases for fixed angles
      }
    }
    else{
      for (uint j=0; j<=halfDiscr; j++){
        COUT(j)
        if(j==0) { 
          matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1); 
        }
        else{
          matrix[i*discr+j]          =DP::Cell(mod2pi(points[i].th()-(j*1.0)*dtheta), l, -1);
          matrix[i*discr+j+halfDiscr]=DP::Cell(mod2pi(points[i].th()+(j*1.0)*dtheta), l, -1); 
        }
      }
      uint j=discr-addedAngles;
      for (std::set<Angle>::iterator it=moreAngles[i].begin(); it!=moreAngles[i].end(); ++it){
        matrix[i*discr+j]=DP::Cell(*it, l, -1);
        j++;
      }
      for (; j<discr; j++){
        matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1);
      }
    }
  }

  cudaMemcpy(dev_matrix, matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyHostToDevice);
  cudaCheckError(cudaGetLastError());

  size_t jump=(params.size()>1 ? params[1] : 3);
  size_t iter=0;
  if ((size-1)%jump==0) { iter=(size-1)/jump; }
  else                  { iter=(size_t)(((size-1)+jump)/jump); }

  size_t totThreads=jump*discr*discr;
  size_t threads=totThreads>nThreads ? nThreads : totThreads;
  size_t blocks=((int)(totThreads/threads)+1)*numberOfSMs; 
  
  real_type *results, *dev_results1, *dev_results2, *dev_resultsapp;
  cudaMallocHost(&results, sizeof(real_type)*jump*discr*discr);
  cudaMalloc(&dev_results1, sizeof(real_type)*jump*discr*discr);
  cudaMalloc(&dev_results2, sizeof(real_type)*jump*discr*discr);
    
  for (int i=iter-1; i>=0; i--){
    computeMore<<<blocks, threads>>>(dev_matrix, dev_results1, dev_fixedAngles, dev_params, dev_points, jump, discr, size, i);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError());
    
    cudaMemcpy(results, dev_results1, sizeof(real_type)*jump*discr*discr, cudaMemcpyDeviceToHost);
    cudaCheckError(cudaGetLastError());

    dev_resultsapp=dev_results1;
    dev_results1=dev_results2;
    dev_results2=dev_resultsapp;

    bestAnglesPerCell(matrix, results, fixedAngles, size, discr, i, jump, nThreads, numberOfSMs);
    cudaMemcpy(dev_matrix, matrix, sizeof(real_type)*size*discr, cudaMemcpyHostToDevice);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError());
    
    #ifdef DEBUG
    printf("\n");
    #endif
  }
#ifdef DEBUG
  //Retrieve angles
  cout << "Computing best angles" << endl;
#endif
  std::pair<LEN_T, std::vector<Angle> > ret=bestAnglesMatrix(matrix, discr, size, fixedAngles);
#ifdef DEBUG
  std::vector<Angle> bestA=ret.second;
  printV(bestA)
#endif
  
#ifdef DEBUG
  LEN_T Length=0.0;
  for (unsigned int i=bestA.size()-1; i>0; i--){
    points[i].th(bestA[i]);
    points[i-1].th(bestA[i-1]);
    CURVE c(points[i-1], points[i], params.data());
    Length+=c.l();
  }
  cout << "\tAllInOne length: " << setprecision(20) << Length << " " << setprecision(12) << (Length-7.467562181965) << endl;

  cout << "Printing for Matlab" << endl;
  cout << "X=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].x() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "Y=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].y() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "th=[";
  for (unsigned int i=0; i<bestA.size(); i++){ cout << bestA[i] << (i!=bestA.size()-1 ? ", " : "];\n"); }
  cout << "KMAX: " << params[0] << endl;
#endif
  cudaFreeHost(matrix);
  cudaFreeHost(results);

  cudaFree(dev_matrix);
  cudaFree(dev_params);
  cudaFree(dev_fixedAngles);
  cudaFree(dev_points);
  cudaFree(dev_results1);
  cudaFree(dev_results2);

  return ret;
}

std::pair<LEN_T, std::vector<Angle> >
DP::solveDP(std::vector<Configuration2>& points, const std::vector<bool> fixedAngles,
            std::vector<real_type> params, int discr, uint nRefs, bool saveAngles,
            short type, uint threads, Angle _fullAngle){
  ////std::cout << "ciao2.1\n";
  if (points.size()!=fixedAngles.size()){
    std::cerr << "Number of points and number of fixed angles are not the same: " << points.size() << "!=" << fixedAngles.size() << std::endl;
    return std::pair<LEN_T, std::vector<Angle> >(MAX_LEN_T, std::vector<Angle>());
  }

  Angle fullAngle=_fullAngle;
  std::pair<LEN_T, std::vector<Angle> > ret;

  for(uint i=0; i<nRefs+1; ++i){
    switch(type){
      case 1:{
        ret=solveDPMatrixAllocator (points, discr, fixedAngles, params, fullAngle, threads, i);
        break;
      }
      case 2: default:{
        ret=solveDPAllIn1          (points, discr, fixedAngles, params, fullAngle, threads, i);
      }
    }
    std::vector<Angle> angles=ret.second;

    if(saveAngles){
      for (uint j=0; j<angles.size(); j++){
        if (!fixedAngles[j]){
          points[j].th(angles[j]);
        }
      }
    }

    if (i==0){
      fullAngle=fullAngle/(discr)*1.5;
      discr++; //This is because, yes.
    }
    else{
      fullAngle=fullAngle/(discr-1)*1.5;
    }
    angles.clear();
  }
  return ret;
}

#endif //CUDA_ON


