
#include <hip/hip_runtime.h>
#ifdef CUDA_ON
#include <dp.cuh>

// returns (up to) two circles through two points, given the radius
// Marco Frego and Paolo Bevilacqua in "An Iterative Dynamic Programming Approach to the Multipoint Markov-Dubins Problem" 2020.
static inline
void circles(real_type x1, real_type y1, real_type x2, real_type y2, real_type r, std::vector<real_type> & XC, std::vector<real_type> & YC) 
{
  real_type TOL = 1e-8;
  
  real_type q = std::hypot(x2-x1, y2-y1);
  real_type x3 = 0.5*(x1+x2);
  real_type y3 = 0.5*(y1+y2);

  real_type delta = r*r-q*q/4.;
    
  XC.clear();
  YC.clear();

  if (delta < -TOL) {
    return;
  }
  
  if (delta < TOL) 
  {
    XC.push_back(x3);
    YC.push_back(y3);
  }
  else
  {
    real_type deltaS = std::sqrt(delta);
    XC.push_back(x3 + deltaS*(y1-y2)/q);
    YC.push_back(y3 + deltaS*(x2-x1)/q);
    XC.push_back(x3 - deltaS*(y1-y2)/q);
    YC.push_back(y3 - deltaS*(x2-x1)/q);
  }
}

// Marco Frego and Paolo Bevilacqua in "An Iterative Dynamic Programming Approach to the Multipoint Markov-Dubins Problem" 2020.
uint DP::guessInitialAngles(std::vector<std::set<Angle> >& moreAngles, const std::vector<Configuration2<real_type> >& points, const std::vector<bool> fixedAngles, const real_type K){
  uint max=0;
  for (uint i=1; i<points.size(); i++){
    //std::cout << points[i-1] << "   " << points[i] << std::endl;
    moreAngles.push_back(std::set<Angle>());
    if (i==1) { moreAngles.push_back(std::set<Angle>()); }
    //First add the lines connecting two points:
    Angle th = std::atan2((points[i].y()-points[i-1].y()), (points[i].x()-points[i-1].x()));
    if (!fixedAngles[i-1]){ moreAngles[i-1].insert(th); }
    if (!fixedAngles[i])  { moreAngles[i].insert(th); }
    
    //Then add the possible angles of the tangents to two possible circles:
    std::vector<real_type> XC, YC;
    circles(points[i-1].x(), points[i-1].y(), points[i].x(), points[i].y(), 1./K, XC, YC);
    //std::cout << "aligned: " << th << std::endl;
    
    //std::cout << "XC: ";
    //for (auto a : XC) { std::cout << a << ", "; }
    //std::cout << std::endl << "YC: ";
    //for (auto a : YC) { std::cout << a << ", "; }
    for (uint j=0; j<XC.size(); j++){
      if (!fixedAngles[i-1]){
        th = std::atan2(points[i-1].y()-YC[j], points[i-1].x()-XC[j]);
        moreAngles[i-1].insert(th+M_PI/2.);
        moreAngles[i-1].insert(th-M_PI/2.);
      }
      if (!fixedAngles[i]){
        th = std::atan2(points[i].y()-YC[j], points[i].x()-XC[j]);
        moreAngles[i].insert(th+M_PI/2.);
        moreAngles[i].insert(th-M_PI/2.);
      }
    }
    if (moreAngles[i-1].size()>max){
      max=moreAngles[i-1].size();
    }
    if (i==points.size()-1 && moreAngles[i].size()>max){
      max=moreAngles[i].size();
    }
    //std::cout << std::endl << std::endl << std::endl;
  }  
  //for (auto v : moreAngles){
  //  for (auto a : v){
  //    std::cout << std::setw(11) << std::setprecision(5) << a*360/(2*M_PI) << " ";
  //  }
  //  std::cout << endl;
  //}

  //std::cout << "points: " << points.size() << " angles: " << moreAngles.size() << std::endl; 
  return max;
}

__global__ void printResults(real_type* results, size_t discr, size_t size){
  for (int i=0; i<size; i++){
    for(int j=0; j<discr; j++){
      for(int h=0; h<discr; h++){
        printf("(%2.0f,%.2f)", (float)((i*discr+j)*discr+h), results[(i*discr+j)*discr+h]);
      }
      printf("\t");
    }
    printf("\n");
  }
}

#endif 

