
#include <hip/hip_runtime.h>
#ifdef CUDA_ON
#include <dp.cuh>

__global__ void dubinsWrapper(Configuration2 c0, Configuration2 c1, double Kmax, double* L){
  CURVE c(c0, c1, Kmax);
  //printf("%.17f\n", c.l());
  L[0]+=c.l();
}

__global__ void printResults(real_type* results, uint discr, uint size){
  for (int i=0; i<size; i++){
    for(int j=0; j<discr; j++){
      for(int h=0; h<discr; h++){
        printf("(%2.0f,%.2f)", (float)((i*discr+j)*discr+h), results[(i*discr+j)*discr+h]);
      }
      printf("\t");
    }
    printf("\n");
  }
}

__global__ void printMatrix(DP::Cell* matrix, uint discr, uint size){
  for (int i=0; i<size; i++){
    for(int j=0; j<discr; j++){
      printf("(%d,%d)", (i*discr+j), matrix[i*discr+j].next());
    }
    printf("\n");
  }
}


// returns (up to) two circles through two points, given the radius
// Marco Frego and Paolo Bevilacqua in "An Iterative Dynamic Programming Approach to the Multipoint Markov-Dubins Problem" 2020.
static inline
void circles(real_type x1, real_type y1, real_type x2, real_type y2, real_type r, std::vector<real_type> & XC, std::vector<real_type> & YC) 
{
  real_type TOL = 1e-8;
  
  real_type q = std::hypot(x2-x1, y2-y1);
  real_type x3 = 0.5*(x1+x2);
  real_type y3 = 0.5*(y1+y2);

  real_type delta = r*r-q*q/4.;
    
  XC.clear();
  YC.clear();

  if (delta < -TOL) {
    return;
  }
  
  if (delta < TOL) 
  {
    XC.push_back(x3);
    YC.push_back(y3);
  }
  else
  {
    real_type deltaS = std::sqrt(delta);
    XC.push_back(x3 + deltaS*(y1-y2)/q);
    YC.push_back(y3 + deltaS*(x2-x1)/q);
    XC.push_back(x3 - deltaS*(y1-y2)/q);
    YC.push_back(y3 - deltaS*(x2-x1)/q);
  }
}

// Marco Frego and Paolo Bevilacqua in "An Iterative Dynamic Programming Approach to the Multipoint Markov-Dubins Problem" 2020.
// The function name is pretty self-explainatory 
uint guessInitialAngles(std::vector<std::set<Angle> >& moreAngles, const std::vector<Configuration2>& points, const std::vector<bool> fixedAngles, const real_type K){
  uint max=0;
  for (uint i=1; i<points.size(); i++){
    moreAngles.push_back(std::set<Angle>());
    if (i==1) { moreAngles.push_back(std::set<Angle>()); }
    //First add the lines connecting two points:
    Angle th = std::atan2((points[i].y()-points[i-1].y()), (points[i].x()-points[i-1].x()));
    if (!fixedAngles[i-1]){ moreAngles[i-1].insert(th); }
    if (!fixedAngles[i])  { moreAngles[i].insert(th); }
    
    //Then add the possible angles of the tangents to two possible circles:
    std::vector<real_type> XC, YC;
    circles(points[i-1].x(), points[i-1].y(), points[i].x(), points[i].y(), 1./K, XC, YC);
    
    for (uint j=0; j<XC.size(); j++){
      if (!fixedAngles[i-1]){
        th = std::atan2(points[i-1].y()-YC[j], points[i-1].x()-XC[j]);
        moreAngles[i-1].insert(th+M_PI/2.);
        moreAngles[i-1].insert(th-M_PI/2.);
      }
      if (!fixedAngles[i]){
        th = std::atan2(points[i].y()-YC[j], points[i].x()-XC[j]);
        moreAngles[i].insert(th+M_PI/2.);
        moreAngles[i].insert(th-M_PI/2.);
      }
    }
    if (moreAngles[i-1].size()>max){
      max=moreAngles[i-1].size();
    }
    if (i==points.size()-1 && moreAngles[i].size()>max){
      max=moreAngles[i].size();
    }
  }  
  return max;
}

std::vector<Angle> bestAngles(DP::Cell* matrix, int discr, int size){
  DP::Cell* best=&matrix[0];
  //Find best path
  for (int i=size; i<discr*size; i+=size){
    if (best->l()>matrix[i].l()  && matrix[i].l()!=0){ //TODO The second check is actually a bug in solveCell, but I'm not in the right mind to find this bug, please fix later
      best=&matrix[i];
    }
  }
  //Retrieve best angles
  vector<Angle> ret(1, best->th());
  uint nextID=best->next();
  while (nextID!=0){
    ret.push_back(matrix[nextID].th());
    nextID=matrix[nextID].next();
  }
  return ret;
}

std::vector<Angle> 
bestAnglesMatrix(DP::Cell* matrix, int discr, int size, const std::vector<bool>& fixedAngles){
  DP::Cell* best=&matrix[0];

  if (!fixedAngles[0]){
    for(int i=1; i<discr; i++){
      if (matrix[i].l()<best->l())
        best=&matrix[i];
    }
  }

  //std::cout << "In function Length: " << std::setw(20) << std::setprecision(17) << best->l() << std::endl;

  std::vector<Angle> ret(1, best->th());
  int nextID=best->next()+discr;
  for (int i=1; i<size; i++){
    ret.push_back(matrix[nextID].th());
    nextID=matrix[nextID].next()+(i+1)*discr;
  }
  return ret;
}

__global__ void solveCol( DP::Cell* matrix, uint discr, uint size, const bool* fixedAngles, 
                          Configuration2 c0, Configuration2 c1, 
                          Angle a00, Angle a01, real_type* params, int i, Angle fullAngle, bool halveDiscr
                        ){
  int tidx=threadIdx.x+blockDim.x*blockIdx.x;
  int stride=blockDim.x*gridDim.x;
  int halfDiscr=(discr-1)/2;
  int j=tidx;

  // if (j<discr){
  for (; j<discr; j+=stride){
    Angle bestA=0.0;
    LEN_T bestL=MAX_LEN_T; 
    int bestK=0;
    if (!fixedAngles[i-1]){ //If angle is fixed I don't have to change it
      double hj=fullAngle*((j-halfDiscr)*1.0)/(((halveDiscr ? halfDiscr : discr)*1.0));
      c0.th(a00+hj); 
    } 
    
    for (int k=0; k<discr; k++){ //SolveCell
      LEN_T currL=MAX_LEN_T;
      if (!fixedAngles[i]){ //If angle is fixed I don't have to change its
        double hk=fullAngle*((k-halfDiscr)*1.0)/(((halveDiscr ? halfDiscr : discr)*1.0));
        c1.th(a01+hk); 
      } 
      CURVE c=CURVE(c0, c1, params); 
      DP::Cell* next=(i==size-1 ? NULL : &matrix[k*size+(i+1)]);
      if (c.l()>0){
        currL=c.l();
        if (next!=NULL){
          currL+=next->l();
        }  
        if (currL<bestL || bestL==MAX_LEN_T){
          bestL=currL;
          bestA=c1.th();
          bestK=k;
        }
      }
      if (fixedAngles[i]){ k=discr; } //If the angle is fixed I don't have to change it
    }
    
    if (bestL!=MAX_LEN_T){
      uint nextID=(i==size-1 ? 0 : bestK*size+(i+1));
      matrix[j*size+i]=DP::Cell(bestA, bestL, nextID);
    }
    if (i==1){
      matrix[size*j]=DP::Cell(c0.th(), bestL, (size*j+i));
    }
    if(fixedAngles[i-1]) j=discr;
  }
}

std::vector<Angle> solveDPFirstVersion (std::vector<Configuration2> points, uint discr, const std::vector<bool> fixedAngles, std::vector<real_type> params, Angle fullAngle=2*M_PI, bool halveDiscr=false, bool guessInitialAnglesVal=false, uint nThreads=0){
  if (points.size()!=fixedAngles.size()){
    cerr << "Number of points and number of fixed angles are not the same: " << points.size() << "!=" << fixedAngles.size() << endl;
    return std::vector<Angle>();
  }
  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());
  
  uint size=points.size();
  discr=(discr%2==0 ?  discr+1 : discr);
  //if (guessInitialAnglesVal){
  //  guessInitialAngles(points, fixedAngles);
  //}
  DP::Cell* matrix=new DP::Cell[size*discr];
  DP::Cell* dev_matrix;
  cudaMalloc(&dev_matrix, sizeof(DP::Cell)*discr*size);
  cudaCheckError(cudaGetLastError());
  
  bool* dev_fixedAngles=cudaSTDVectorToArray<bool>(fixedAngles);
  real_type* dev_params=cudaSTDVectorToArray<real_type>(params);
  
  cudaCheckError(cudaGetLastError());
  
  for (int i=size-1; i>0; i--){
    Configuration2 c0=points[i-1];
    Configuration2 c1=points[i];
    Angle a00=c0.th(), a01=c1.th();
    size_t threads=discr>nThreads ? nThreads : discr;
    size_t blocks=numberOfSMs; 
    // size_t blocks=((int)(discr/threads)+1)*numberOfSMs; 
    if(fixedAngles[i-1]){
      threads=1;
      blocks=1;
    }
    solveCol<<<blocks, threads>>>(dev_matrix, discr, size, dev_fixedAngles, c0, c1, a00, a01, dev_params, i, fullAngle, halveDiscr);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError());
  }

  cudaMemcpy(matrix, dev_matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyDeviceToHost);
  cudaCheckError(cudaGetLastError());

#ifdef DEBUG
  cout << "Printing " << endl;
  printVM(matrix, discr, size)
  //Retrieve angles
  cout << "Computing best angles" << endl;
#endif
  std::vector<Angle> bestA=bestAngles(matrix, discr, size);
#ifdef DEBUG
  printV(bestA)
#endif
  
#ifdef DEBUG
  LEN_T Length=0.0;
  for (unsigned int i=bestA.size()-1; i>0; i--){
    points[i].th(bestA[i]);
    points[i-1].th(bestA[i-1]);
    CURVE c(points[i-1], points[i], params.data());
    Length+=c.l();
  }
  cout << "Length: " << setprecision(20) << Length << endl;

  cout << "Printing for Matlab" << endl;
  cout << "X=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].x() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "Y=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].y() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "th=[";
  for (unsigned int i=0; i<bestA.size(); i++){ cout << bestA[i] << (i!=bestA.size()-1 ? ", " : "];\n"); }
  cout << "KMAX: " << params[0] << endl;
#endif

  delete matrix;
  cudaFree(dev_matrix);
  cudaFree(dev_fixedAngles);
  cudaFree(dev_params);

  return bestA;
}


__global__ void solveMatrixCol (DP::Cell* matrix, uint discr, uint size, const bool* fixedAngles, 
                                Configuration2 c0, Configuration2 c1, 
                                real_type* params, int i, uint ref=0){
  uint tidx=threadIdx.x+blockDim.x*blockIdx.x;
  uint stride=blockDim.x*gridDim.x;

  uint j=tidx;
  // if (j<discr){
  for (; j<discr; j+=stride){
    c0.th(matrix[i*discr+j].th());
    for (int h=0; h<(int)(discr); h++){
      c1.th(matrix[(i+1)*discr+h].th());

      CURVE c=CURVE(c0, c1, params);
      LEN_T currL=c.l()+matrix[(i+1)*discr+h].l();
      //if (ref==3 && i==0 && j==0){
      //  printf("x0: %.2f y0: %.2f th0: %.16f x1: %.2f y1: %.2f th1: %.16f matrix[i*discr+j].l(): %.16f currL %.16f c.l(): %.16f matrix[(i+1)*discr+h].l(): %.16f\n", c0.x(), c0.y(), c0.th(), c1.x(), c1.y(), c1.th(), (matrix[i*discr+j].l()<10000.0 ? matrix[i*discr+j].l() : 10000.0), currL, c.l(), matrix[(i+1)*discr+h].l());
      //}
      if (currL<matrix[i*discr+j].l()) {
        matrix[i*discr+j].l(currL);
        //printf("nextID in func: %u %d\n", h, h);
        matrix[i*discr+j].next(h);
      }
      if (fixedAngles[i+1]) {h=discr;}
    }
    if (matrix[i*discr+j].next()==-1) printf("[%u] BIG NO\n", i*discr+j);
    if (fixedAngles[i]) {j=discr;}
  }
}

void solveDPMatrix (std::vector<Configuration2> points, DP::Cell* dev_matrix, uint discr, std::vector<bool> fixedAngles, 
                    bool* dev_fixedAngles, real_type* dev_params, uint nThreads=128, uint ref=0){

  //REMOVE
  size_t size=points.size();
  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());


  for (int i=size-2; i>=0; i--){
    Configuration2 c0=points[i];
    Configuration2 c1=points[i+1];

    size_t threads=discr>nThreads ? nThreads : discr;
    size_t blocks=((int)(discr/threads)+1)*numberOfSMs; 
    
    //size_t nBlocksGivenThreads=(int)(discr/threads);
    //size_t blocks=1;
    //if (nBlocksGivenThreads>0 && nBlocksGivenThreads<numberOfSMs){
    //  blocks=nBlocksGivenThreads;
    //}
    //else if (nBlocksGivenThreads==0){
    //  blocks=1;
    //}
    //else{
    //  blocks=((int)(nBlocksGivenThreads/numberOfSMs))*numberOfSMs;
    //}
    if(fixedAngles[i]){
      threads=1;
      blocks=1;
    }
    solveMatrixCol<<<blocks, threads>>>(dev_matrix, discr, size, dev_fixedAngles, c0, c1, dev_params, i, ref);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError()); 
  }
  if (ref==30){
    printMatrix<<<1,1>>>(dev_matrix, discr, size);
    cudaDeviceSynchronize();
  }
}

std::vector<Angle> solveDPMatrixAllocator (std::vector<Configuration2> points, uint discr, const std::vector<bool> fixedAngles, std::vector<real_type> params, Angle fullAngle=2*M_PI, bool halveDiscr=false, bool guessInitialAnglesVal=false, uint nThreads=0, uint ref=0){
  size_t size=points.size();
  DP::Cell* matrix;
  bool* dev_fixedAngles=cudaSTDVectorToArray<bool>(fixedAngles);
  real_type* dev_params=cudaSTDVectorToArray<real_type>(params);  
  DP::Cell* dev_matrix;
  
  if (points.size()!=fixedAngles.size()){
    cerr << "Number of points and number of fixed angles are not the same: " << points.size() << "!=" << fixedAngles.size() << endl;
    return std::vector<Angle>();
  }
  
  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());
  
  std::vector<std::set<Angle> > moreAngles;
  uint addedAngles=0;
  if(guessInitialAnglesVal){
    addedAngles=guessInitialAngles(moreAngles, points, fixedAngles, params[0]);
  }
  
  uint halfDiscr=(uint)((discr-(discr%2==0 ? 0 : 1))/2);
  real_type dtheta=fullAngle/((halveDiscr ? (int)(discr/2) : discr)*1.0);
  discr+=addedAngles;
  cudaMallocHost(&matrix, (sizeof(DP::Cell)*size*discr));
  cudaMalloc(&dev_matrix, sizeof(DP::Cell)*size*discr);
   
  
  //std::cout << "halveDiscr: " << (halveDiscr==true ? "true" : "false") << std::endl; 
  //std::cout << "discr: " << discr << std::endl;
  //std::cout << "size: " << size << std::endl;
  //std::cout << "hrange: " << std::setw(20) << std::setprecision(17) << fullAngle << std::endl;
  //std::cout << "dtheta: " << std::setw(20) << std::setprecision(17) << dtheta << std::endl;
  //std::cout << "hn: " << halfDiscr << std::endl;
  //std::cout << "pippo: " << matrix[discr].th()-fullAngle/2.0 << std::endl;

  for (uint i=0; i<size; i++){ //TODO change back, remove l=-1 if fixedAngles
    LEN_T l = (i==size-1 ? 0 : std::numeric_limits<LEN_T>::max());
    for (uint j=0; j<=halfDiscr; j++){
      if (fixedAngles[i]){
        matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1);
        break;
      }
      else {
        if(j==0) { 
          matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1); 
        }
        else{
          matrix[i*discr+j]          =DP::Cell(mod2pi(points[i].th()-(j*1.0)*dtheta), l, -1);
          matrix[i*discr+j+halfDiscr]=DP::Cell(mod2pi(points[i].th()+(j*1.0)*dtheta), l, -1); 
        }
      }
    }
    if (guessInitialAnglesVal){
      uint j=discr-addedAngles;
      if (!fixedAngles[i]){
        for (std::set<Angle>::iterator it=moreAngles[i].begin(); it!=moreAngles[i].end(); ++it){
          matrix[i*discr+j]=DP::Cell(*it, l, -1);
          j++;
        }
        for (; j<discr; j++){
          matrix[i*discr+j]=DP::Cell(0, l, -1);
        }
      }
    }
  }
    
  cudaMemcpy(dev_matrix, matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyHostToDevice);
  cudaCheckError(cudaGetLastError());

  solveDPMatrix(points, dev_matrix, discr, fixedAngles, dev_fixedAngles, dev_params, nThreads, ref);

  cudaMemcpy(matrix, dev_matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyDeviceToHost);
  cudaCheckError(cudaGetLastError());


  //if (ref==8){
  //  cout << "Printing " << endl;
  //  printVM(matrix, size, discr)
  //}
#ifdef DEBUG
  //Retrieve angles
  cout << "Computing best angles" << endl;
#endif
  std::vector<Angle> bestA=bestAnglesMatrix(matrix, discr, size, fixedAngles);
#ifdef DEBUG
  printV(bestA)
#endif
  
#ifdef DEBUG
  LEN_T Length=0.0;
  for (unsigned int i=bestA.size()-1; i>0; i--){
    points[i].th(bestA[i]);
    points[i-1].th(bestA[i-1]);
    CURVE c(points[i-1], points[i], params.data());
    Length+=c.l();
  }
  cout << "\tMatrix length: " << setprecision(20) << Length << " " << setprecision(12) << (Length-7.46756219733842652175326293218) << endl;
  cout << "Printing for Matlab" << endl;
  cout << "X=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].x() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "Y=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].y() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "th=[";
  for (unsigned int i=0; i<bestA.size(); i++){ cout << bestA[i] << (i!=bestA.size()-1 ? ", " : "];\n"); }
  cout << "KMAX: " << params[0] << endl;
#endif

  cudaFreeHost(matrix);

  cudaFree(dev_matrix);
  cudaFree(dev_params);
  cudaFree(dev_fixedAngles);

  return bestA;
}


__global__ void computeMore(DP::Cell* matrix, real_type* results, const bool* fixedAngles,
                            real_type* params, const Configuration2* points, 
                            size_t jump, size_t discr, size_t size, size_t iter){ //TODO It may be possible to remove cmp and use iter and i to compute the position
  uint tidx=threadIdx.x+blockDim.x*blockIdx.x;

  uint j=tidx;
  if (j<discr*jump*discr){        //j must be less than the number of rows (jump) times the number of inner cells per cell, times the number of cells per row
    uint cell=(int)(tidx/discr);  //The big cell
    uint inCell=tidx%discr;       //The small cell insider the big cell
    uint cmpId=(int)(cell/discr); //The row in this call
    uint pos=iter*jump+cmpId;     //The row w.r.t. the whole matrix

    if (pos<size-1){ 
      Configuration2 c0=points[pos];
      Configuration2 c1=points[pos+1];

      if (!fixedAngles[pos])   {c0.th(matrix[cell+iter*jump*discr].th());}
      if (!fixedAngles[pos+1]) {c1.th(matrix[inCell+(pos+1)*discr].th());}

      CURVE c=CURVE(c0, c1, params);
      if (c.l()>0){
        results[cell*discr+inCell]=c.l();
      }
    }
  }
}


void bestAnglesPerCell( DP::Cell* matrix, real_type* results, const std::vector<bool> fixedAngles, 
                        size_t size, size_t discr, size_t iter, size_t jump, size_t _threads, size_t numberOfSMs){
  //MIND THAT WE START FROM THE LAST ROW OF THE MATRIX
  int startRowIDM=iter*jump;        //Given #jump rows, this is the id of the first row in the jump group
  uint lastRowIDM=iter*jump+jump-1;  //Given #jump rows, this is the id of the last row in the jump group
  lastRowIDM=(lastRowIDM<size ? lastRowIDM : size-1);
  for (int i=lastRowIDM; i>=startRowIDM; i--){ //Cycle through the rows
    if (i==(int)(size-1)){continue;}    //If it's the last row, then skip it.
    uint startCellM=i*discr;              //Given #jump rows, this is the id of the cell in the row in the jump group I'm considering
    //std::cout << "startRowIDM: " << startRowIDM << std::endl;
    //std::cout << "lastRowIDM: " << lastRowIDM << std::endl;
    //std::cout << "startCellM: " << startCellM << std::endl;
    //std::cout << "i: " << i << std::endl;
    for (uint cellIDM=startCellM; cellIDM<startCellM+discr; cellIDM++){ //Cycle through all the cells in the row  
      for (uint h=0; h<discr; h++){        //Each cell in matrix corresponds to discr cells in results. Each h is in results is the same as the next row
        uint cellIDR=cellIDM*discr+h-startRowIDM*discr*discr;
        double currL=results[cellIDR]+matrix[(i+1)*discr+h].l();
        // int a=-2;
        if(currL<matrix[cellIDM].l()){
          matrix[cellIDM].l(currL);
          // a=matrix[cellIDM].next(h);
          //std::cout << "a: " << a << std::endl;
        }
        //if (cellIDM>29 && cellIDM<45){
        //  std::cout << "i: " << i << std::endl;
        //  std::cout << "cellIDM: " << cellIDM << std::endl;
        //  std::cout << "cellIDR: " << cellIDR << std::endl;
        //  std::cout << "results[cellIDR]: " << results[cellIDR] << std::endl;
        //  std::cout << "currL: " << currL << std::endl;
        //  std::cout << "matrix[cellIDM].l(): " << matrix[cellIDM].l() << std::endl;
        //  std::cout << "a: " << a << std::endl;
        //  std::cout << "matrix[cellIDM].n(): " << matrix[cellIDM].next() << std::endl;
        //}
        //std::cout << "matrix[(i+1)*discr+h].l(): " << matrix[(i+1)*discr+h].l() << std::endl;
        if (fixedAngles[i+1]){ h=discr; }
      } 
      if (matrix[cellIDM].next()<0) {printf("[%u] BIG NO\n", cellIDM);}
    }
    if (fixedAngles[i]){ i=startRowIDM-1; }
  }
}

std::vector<Angle> 
solveDPAllIn1 ( std::vector<Configuration2> points, 
                uint discr, const std::vector<bool> fixedAngles, 
                std::vector<real_type> params, Angle fullAngle, 
                bool halveDiscr=false, bool guessInitialAnglesVal=false, 
                uint nThreads=0, uint ref=0){
  if (points.size()!=fixedAngles.size()){
    cerr << "Number of points and number of fixed angles are not the same: " << points.size() << "!=" << fixedAngles.size() << endl;
    return std::vector<Angle>();
  }
  
  int numberOfSMs; cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, cudaGetdeviceID());

  uint addedAngles=0;
  std::vector<std::set<Angle> > moreAngles;
  //guessInitialAnglesVal=false;
  if(guessInitialAnglesVal){
    addedAngles=guessInitialAngles(moreAngles, points, fixedAngles, params[0]);
  }
  
  size_t size=points.size();
  //discr=(discr%2==0 ? discr+1 : discr); //So.... since we add always the angle in position 0, we'll always have an odd number of discretizionations... 
  uint halfDiscr=(uint)(discr/2);
  real_type dtheta=fullAngle/((halveDiscr ? (int)(discr/2) : discr)*1.0);
  discr+=addedAngles;

  DP::Cell* matrix;
  cudaMallocHost(&matrix, sizeof(DP::Cell)*size*(discr));
  DP::Cell* dev_matrix;
  cudaMalloc(&dev_matrix, sizeof(DP::Cell)*size*(discr));
  //real_type* results;
  //cudaMallocHost(&results, sizeof(real_type)*size*discr*discr);
  
  bool* dev_fixedAngles=cudaSTDVectorToArray<bool>(fixedAngles);
  real_type* dev_params=cudaSTDVectorToArray<real_type>(params);  
  Configuration2* dev_points=cudaSTDVectorToArray<Configuration2>(points);
  
  //std::cout << "halveDiscr: " << (halveDiscr==true ? "true" : "false") << std::endl; 
  //std::cout << "discr: " << discr << std::endl;
  //std::cout << "halfDiscr: " << halfDiscr << std::endl;
  //std::cout << "hrange: " << std::setw(20) << std::setprecision(17) << fullAngle << std::endl;
  //std::cout << "dtheta: " << std::setw(20) << std::setprecision(17) << dtheta << std::endl;
  //std::cout << "hn: " << halfDiscr << std::endl;

  for (uint i=0; i<size; i++){
    LEN_T l = (i==size-1 ? 0 : std::numeric_limits<LEN_T>::max());
    if (fixedAngles[i]){
      for (uint j=0; j<discr; j++){
        matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1);
        //In this case I need to have the row full of the same values. Otherwise I should change the kernel function and add particular cases for fixed angles
      }
    }
    else{
      for (uint j=0; j<=halfDiscr; j++){
        COUT(j)
        if(j==0) { matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1); 
        }
        else{
          matrix[i*discr+j]          =DP::Cell(mod2pi(points[i].th()-(j*1.0)*dtheta), l, -1);
          matrix[i*discr+j+halfDiscr]=DP::Cell(mod2pi(points[i].th()+(j*1.0)*dtheta), l, -1); 
        }
      }
      if (guessInitialAnglesVal){
        uint j=discr-addedAngles;
        if (!fixedAngles[i]){
          for (std::set<Angle>::iterator it=moreAngles[i].begin(); it!=moreAngles[i].end(); ++it){
            matrix[i*discr+j]=DP::Cell(*it, l, -1);
            j++;
          }
        }
        for (; j<discr; j++){
          matrix[i*discr+j]=DP::Cell(points[i].th(), l, -1);
        }
      }
    }
  }

  cudaMemcpy(dev_matrix, matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyHostToDevice);
  cudaCheckError(cudaGetLastError());

  size_t jump=(params.size()>1 ? params[1] : 3);
  size_t iter=0;
  if ((size-1)%jump==0) { iter=(size-1)/jump; }
  else                  { iter=(size_t)(((size-1)+jump)/jump); }

  size_t totThreads=jump*discr*discr;
  size_t threads=totThreads>nThreads ? nThreads : totThreads;
  size_t blocks=((int)(totThreads/threads)+1)*numberOfSMs; 
  
  real_type *results, *dev_results1, *dev_results2, *dev_resultsapp;
  cudaMallocHost(&results, sizeof(real_type)*jump*discr*discr);
  cudaMalloc(&dev_results1, sizeof(real_type)*jump*discr*discr);
  cudaMalloc(&dev_results2, sizeof(real_type)*jump*discr*discr);
  //std::cout << "Iter: " << iter << std::endl;
  //std::cout << "discr: " << discr << std::endl;
    
  for (int i=iter-1; i>=0; i--){  
    //cout << "computing: " << i << endl;
    computeMore<<<blocks, threads>>>(dev_matrix, dev_results1, dev_fixedAngles, dev_params, dev_points, jump, discr, size, i);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError());
    //cout << "<<<<<<<<<<<<<<<<<<<<<<<<" << endl;
    //printResults<<<1, 1>>>(dev_results1, discr, jump);
    //cudaDeviceSynchronize();
    //cout << "<<<<<<<<<<<<<<<<<<<<<<<<" << endl;
    
    cudaMemcpy(results, dev_results1, sizeof(real_type)*jump*discr*discr, cudaMemcpyDeviceToHost);
    cudaCheckError(cudaGetLastError());

    //for (int j=0; j<size; j++){
    //  for (int h=0; h<discr; h++){
    //    for (int k=0; k<discr; k++){
    //      std::cout << "(" << std::setw(2) << (j*discr+h)*discr+k << "," << std::setw(3) << results[(j*discr+h)*discr+k] << ")";
    //    }
    //    std::cout << "\t";
    //  }
    //  std::cout << std::endl;
    //}
    
    dev_resultsapp=dev_results1;
    dev_results1=dev_results2;
    dev_results2=dev_resultsapp;

    bestAnglesPerCell(matrix, results, fixedAngles, size, discr, i, jump, nThreads, numberOfSMs);
    cudaMemcpy(dev_matrix, matrix, sizeof(real_type)*size*discr, cudaMemcpyHostToDevice);
//    bestAnglesPerCell<<<blocks, threads>>>(dev_matrix, dev_results2, dev_fixedAngles, size, discr, i, jump);
//
    //printMatrix<<<1, 1>>>(matrix, discr, size);
    cudaDeviceSynchronize();
    cudaCheckError(cudaGetLastError());
    
    #ifdef DEBUG
    printf("\n");
    #endif
  }
  //cudaMemcpy(matrix, dev_matrix, sizeof(DP::Cell)*size*discr, cudaMemcpyDeviceToHost);
  //cudaCheckError(cudaGetLastError());
  //for(int i=0; i<size; i++){
  //  for(int j=0; j<discr; j++){
  //    printf("%5d ", matrix[i*discr+j].next()); 
  //  }
  //  std::cout << std::endl;
  //}

  //if (ref==8){
  //  cout << "Printing " << endl;
  //  printVM(matrix, size, discr)
  //}
#ifdef DEBUG
  //Retrieve angles
  cout << "Computing best angles" << endl;
#endif
  std::vector<Angle> bestA=bestAnglesMatrix(matrix, discr, size, fixedAngles);
#ifdef DEBUG
  printV(bestA)
#endif
  
#ifdef DEBUG
  LEN_T Length=0.0;
  for (unsigned int i=bestA.size()-1; i>0; i--){
    points[i].th(bestA[i]);
    points[i-1].th(bestA[i-1]);
    CURVE c(points[i-1], points[i], params.data());
    Length+=c.l();
  }
  cout << "\tAllInOne length: " << setprecision(20) << Length << " " << setprecision(12) << (Length-7.467562181965) << endl;

  cout << "Printing for Matlab" << endl;
  cout << "X=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].x() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "Y=[";
  for (unsigned int i=0; i<points.size(); i++){ cout << points[i].y() << (i!=points.size()-1 ? ", " : "];\n"); }
  cout << "th=[";
  for (unsigned int i=0; i<bestA.size(); i++){ cout << bestA[i] << (i!=bestA.size()-1 ? ", " : "];\n"); }
  cout << "KMAX: " << params[0] << endl;
#endif
  cudaFreeHost(matrix);
  cudaFreeHost(results);

  cudaFree(dev_matrix);
  cudaFree(dev_params);
  cudaFree(dev_fixedAngles);
  cudaFree(dev_points);
  cudaFree(dev_results1);
  cudaFree(dev_results2);

  return bestA;
}

std::vector<Angle> DP::solveDP(std::vector<Configuration2>& points, int discr, const std::vector<bool> fixedAngles, std::vector<real_type> params, short type, bool guessInitialAnglesVal, uint nIter, uint threads, Angle _fullAngle){
  Angle fullAngle=_fullAngle;
  std::vector<Angle> angles; 
  //Passing the functions as pointers doesn't work for reasons I don't know
  //std::vector<Angle>(*func)(std::vector<Configuration2> points, uint discr, const std::vector<bool> fixedAngles, std::vector<real_type> params, Angle fullAngle, bool halveDiscr, bool guessInitialAnglesVal)=NULL;
  for(uint i=0; i<nIter+1; ++i){
    //std::cout << "Refinement: " << i << std::endl;
    //std::cout << std::endl;
    switch(type){
      case 0: {
        angles=solveDPFirstVersion(points, discr, fixedAngles, params, fullAngle, (i==0 ? false : true), guessInitialAnglesVal, threads);
        break;
      }
      case 1:{
        angles=solveDPMatrixAllocator(points, discr, fixedAngles, params, fullAngle, (i==0 ? false : true), guessInitialAnglesVal, threads, i);
        break;
      }
      case 2: default:{
        angles=solveDPAllIn1(points, discr, fixedAngles, params, fullAngle, (i==0 ? false : true), guessInitialAnglesVal, threads, i);
      }
    }

    for (uint j=0; j<angles.size(); j++){
      if (!fixedAngles[j]){
        points[j].th(angles[j]);
      }
    }
    //std::cout << "< ";
    //for (auto v : angles){
    //std::cout << std::setw(20) << std::setprecision(17) << mod2pi(v) << " ";
    //}
    //std::cout << ">" << std::endl;
    //
    //LEN_T* Length;
    //cudaMallocManaged(&Length, sizeof(LEN_T));
    //for (unsigned int h=points.size()-1; h>0; ){
    //  dubinsWrapper<<<1,1>>>(points[h-1], points[h], params[0], Length);
    //  cudaDeviceSynchronize();
    //  //std::cout << std::setw(20) << std::setprecision(17) << c.l() << std::endl;
    //  //Length+=c.l();
    //  h--;
    //}
    //std::cout << "Length: " << std::setw(20) << std::setprecision(17) << Length[0] << " " << std::endl; // setprecision(20) << (ABS<real_type>(Length, dLen)) << endl;
    //cudaFree(Length);

    if (i==0){
      fullAngle=fullAngle/(discr)*1.5;
      discr++; //This is because, yes.
    }
    else{
      fullAngle=fullAngle/(discr-1)*1.5;
    }
  }
  return angles;
}

#endif //CUDA_ON


