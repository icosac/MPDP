#include<iostream>
#include<math.h>
using namespace std;

//#define DEBUG

#include<utils.cuh>
#include<dubins.cuh>
#include<dp.cuh>
#include<timeperf.hh>

#include<tests.hh>

vector<vector<Configuration2<double> > > Tests = {
  kaya1, kaya2, kaya3, kaya4, omega, albert
};

vector<K_T> Ks = {3.0, 3.0, 5.0, 3.0, 3.0, 0.1};
vector<uint> discrs = {4, 120, 360, 720, 2000};

#define DISCR 2000

int main (){
  cout << "CUDA" << endl;
  hipFree(0);
#if true
  for (uint discr : discrs){
    cout << "Discr: " << discr << endl;
    for (uint j=0; j<Tests.size(); j++){
      std::vector<bool> fixedAngles;
      vector<Configuration2<double> > v=Tests[j];
      for (int i=0; i<v.size(); i++){
        if (i==0 || i==v.size()-1) {
          fixedAngles.push_back(true);
        }
        else {
          fixedAngles.push_back(false);
        }
      }
      std::vector<real_type> curveParamV={Ks[j]};
      real_type* curveParam=curveParamV.data();

      TimePerf tp;
      tp.start();
      cout << "\t";
      DP::solveDP<Dubins<double> >(v, discr, fixedAngles, curveParamV, false);
      auto time=tp.getTime();
      cout << "\tExample " << j+1 << " completed in " << time << " ms" << endl;
    }
  }
  
#else
  #define KAYA albert
  std::vector<bool> fixedAngles;
  for (int i=0; i<KAYA.size(); i++){
    if (i==0 || i==KAYA.size()-1) {
      fixedAngles.push_back(true);
    }
    else {
      fixedAngles.push_back(false);
    }
  }
  std::vector<real_type> curveParamV={0.1};
  real_type* curveParam=curveParamV.data();

  DP::solveDP<Dubins<double> >(KAYA, DISCR, fixedAngles, curveParamV, true);
#endif
  return 0;
}

