#include "hip/hip_runtime.h"
/**
 * @file main.cu
 * @author Enrico Saccon <enricosaccon96@gmail.com>
 * @license This project is released under the GNU Public License 3.0.
 * @copyright Copyright 2020 Enrico Saccon. All rights reserved.
 * @brief Main file for the Dubins and Reed-Shepp paths computation using CUDA.
 */

#include <iostream>
#include <fstream>
#include <string>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>

#include <utils.cuh>
#include <dubins.cuh>
#include <dp.cuh>
#include <timeperf.hh>
#include <utilities.hh>

#include <tests.hh>

std::vector<Configuration2> example1 = {
		Configuration2 (0, 0, -2.0 * M_PI / 8.0), Configuration2 (2, 2, ANGLE::FREE),
		Configuration2 (6, -1, ANGLE::FREE), Configuration2 (8, 1, 2.0 * M_PI / 8.0)};

std::vector<std::string> testsNames = {"Kaya Example 1", "Kaya Example 2",
																			 "Kaya Example 3", "Kaya Example 4",
																			 "Omega",					 "Circuit"};

std::vector<std::vector<Configuration2>> Tests = {kaya1, kaya2, kaya3, kaya4, omega, spa};

std::vector<K_T> Ks								= {3.0, 3.0, 5.0, 3.0, 3.0, 3.0};
std::vector<uint> discrs					= {4, 16, 90, 360};
std::vector<uint> refins					= {1, 2, 4, 8, 16};
std::vector<LEN_T> exampleLenghts = {
		3.41557885807514871601142658619,
		6.27803455030931356617429628386,
		11.9162126542854860389297755319,
		7.46756219733842652175326293218,
		41.0725016438839318766440555919,
		6988.66098639942993031581863761};	 // the last length is SPA

std::string
nameTest (std::string name, std::string add = "", std::string conc = " ")
{
	if (add == "") { return name; }
	else { return name + conc + add; }
}

__global__ void
dubinsL (Configuration2 c0, Configuration2 c1, real_type k, real_type* L)
{
	Dubins dubins (c0, c1, k);
	L[0] += dubins.l();
}

int
main (int argc, char* argv[])
{
	hipFree (0);

	int devicesCount;
	hipGetDeviceCount (&devicesCount);
	hipDeviceProp_t deviceProperties;
	hipGetDeviceProperties (&deviceProperties, 0);

	std::cout << "Running CUDA" << std::endl;

	if (argc == 1)
	{
		for (int testID = 0; testID < 6; testID++)
		{
			// if (testID!=3){continue;}
			real_type dLen = exampleLenghts[testID];

			std::vector<bool> fixedAngles;
			for (uint i = 0; i < Tests[testID].size(); i++)
			{
				if (i == 0 || i == Tests[testID].size() - 1) { fixedAngles.push_back (true); }
				else { fixedAngles.push_back (false); }
			}
			std::vector<real_type> curveParamV = {Ks[testID], 3};
			real_type* curveParam							 = curveParamV.data();

			for (auto DISCR : discrs)
			{
				if (DISCR != 360) { continue; }
				for (auto r : refins)
				{
					// if (r!=16){continue;}
					TimePerf tp, tp1;
					std::vector<Configuration2> points = Tests[testID];

					tp.start();
					LEN_T Length =
							DP::solveDP (points, fixedAngles, curveParamV, DISCR, r, true, 2).first;
					auto time1 = tp.getTime();

					LEN_T* Length1;
					hipMallocManaged (&Length1, sizeof (LEN_T));
					for (unsigned int idjijij = points.size() - 1; idjijij > 0; idjijij--)
					{
						dubinsL<<<1, 1>>> (points[idjijij - 1], points[idjijij], Ks[testID], Length1);
						hipDeviceSynchronize();

						Dubins c (points[idjijij - 1], points[idjijij], Ks[testID]);
						Length += c.l();
					}

					printf ("%3d & %2d & ", DISCR, r);
					PrintScientific2D ((Length - exampleLenghts[testID]) * 1000.0);
					// printf(" & ");
					// PrintScientific2D((Length1[0]-Length)*1000.0);
					// printf(" & ");
					// PrintScientific2D((Length1[0]-exampleLenghts[testID])*1000.0);
					printf (" & ");
					PrintScientific1D (time1);
					// printf("&%.16f", Length);
					// printf("&%.16f\\\\\n", Length1[0]);
					printf ("\\\\\n");

					hipFree (Length1);
				}
			}
			printf ("\n\n\n\n");
		}
	}
	return 0;
}
